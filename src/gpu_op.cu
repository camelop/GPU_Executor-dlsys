#include "hip/hip_runtime.h"
#include "./c_runtime_api.h"
#include <cassert>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

/* TODO: Your code here */
/* all your GPU kernel code, e.g. matrix_softmax_cross_entropy_kernel */
__global__ void reduce_sum_axis_zero_kernel(const float* input, float* output, int length, int size) {
        int y = blockIdx.x * blockDim.x + threadIdx.x;
        int sum = 0;
        int upper = size * length;
        for (int i=0; i<upper; i+=length) sum += input[y+i];
        output[y] = sum;
}

__global__ void broadcast_to_kernel(const float* input, float* output, int length) {
        int y = blockIdx.x * blockDim.x + threadIdx.x;
        int val = input[y];
        int s = y * length;
        for (int i=0; i<length; i++) output[s+i] = val;
}

__global__ void array_set_kernel(int size, float* array, float value) {
        int y = blockIdx.x * blockDim.x + threadIdx.x;
        array[y] = value;
}

__global__ void matrix_softmax_kernel(int nrow, int ncol,
                                      const float* input,
                                      float* output) {
        int y = blockIdx.x * blockDim.x + threadIdx.x;
        if (y >= nrow) return;
        input += y * ncol;
        output += y * ncol;
        float maxval = *input;
        for (int x=1; x < ncol; ++x) maxval = max(maxval, input[x]);
        float sum = 0;
        for (int x = 0; x < ncol; ++x) sum += exp(input[x] - maxval);
        for (int x = 0; x < ncol; ++x) output[x] = exp(input[x]) / sum;
}


// y = inputs[0], y_ = inputs[1]
// np.mean(-np.sum(y_ * np.log(softmax(y)), axis=1), keepdims=True)
__global__ void matrix_softmax_cross_entropy_kernel(int nrow, int ncol,
                                                    const float *input_a,
                                                    const float *input_b,
                                                    float *output) {
        // Dynamic shared memory, size provided at kernel launch.
        extern __shared__ float loss_per_row[];
        // Two dimensional thread blocks.
        int y = blockIdx.x * blockDim.x + threadIdx.x;
        if (y >= nrow) {
                return;
        }
        input_a += y * ncol;
        input_b += y * ncol;
        float maxval = *input_a;
        // Find max for a row.
        for (int x = 1; x < ncol; ++x) {
                maxval = max(maxval, input_a[x]);
        }
        // Deduct by max for a row, and raise to exp.
        float sum = 0;
        for (int x = 0; x < ncol; ++x) {
                sum += exp(input_a[x] - maxval);
        }
        // Compute per-row loss.
        float loss = 0;
        for (int x = 0; x < ncol; ++x) {
                loss -= input_b[x] * log(exp(input_a[x] - maxval) / sum);
        }
        loss_per_row[y] = loss;
        __syncthreads();
        // Compute reduce_mean across rows.
        float mean_loss = 0;
        // Use a single thread to reduce mean across rows.
        if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
                for (int i = 0; i < nrow; ++i) {
                        mean_loss += loss_per_row[i];
                }
                mean_loss /= nrow;
                output[0] = mean_loss;
        }
}

int DLGpuArraySet(DLArrayHandle arr, float value) { /* TODO: Your code here */
        int size = arr->shape[0];
        float *array = (float *)arr->data;
        float val = value;
        dim3 threads;
        if (size <= 1024) {
                threads.x = size;
        } else {
                if (size < 1024 * 1024) {
                        threads.x = 1024;
                        threads.y = size / 1024;
                } else {
                        threads.x = 1024;
                        threads.y = 1024;
                        threads.z = size / 1024 / 1024;
                }
        }
        array_set_kernel<<<1, threads >>>(size, array, val);
        return 0;
}

int DLGpuBroadcastTo(const DLArrayHandle input, DLArrayHandle output) {
        int size = input->shape[0];
        int ndim = input->ndim;
        int length = 1;
        for (int i=1; i<ndim; i++) length*=input->shape[i];
        const float* input_data = (const float*) input->data;
        float* output_data = (float*) output->data;
        dim3 threads;
        threads.x = nrow % 1024;
        int nblocks = nrow / 1024;
        broadcast_to_kernel <<< nblocks, threads >>> (input_data, output_data, length);
        return 0;
}

int DLGpuReduceSumAxisZero(const DLArrayHandle input, DLArrayHandle output) {
        int size = input->shape[0];
        int ndim = input->ndim;
        int length = 1;
        for (int i=1; i<ndim; i++) length*=input->shape[i];
        const float* input_data = (const float*) input->data;
        float* output_data = (float*) output->data;
        dim3 threads;
        threads.x = length % 1024;
        int nblocks = length / 1024;
        reduce_sum_axis_zero_kernel <<< nblocks, threads >>> (input_data, output_data, length, size);
        return 0;
}

int DLGpuMatrixElementwiseAdd(const DLArrayHandle matA,
                              const DLArrayHandle matB, DLArrayHandle output) {
        /* TODO: Your code here */
        return 0;
}

int DLGpuMatrixElementwiseAddByConst(const DLArrayHandle input, float val,
                                     DLArrayHandle output) {
        /* TODO: Your code here */
        return 0;
}

int DLGpuMatrixElementwiseMultiply(const DLArrayHandle matA,
                                   const DLArrayHandle matB,
                                   DLArrayHandle output) {
        /* TODO: Your code here */
        return 0;
}

int DLGpuMatrixMultiplyByConst(const DLArrayHandle input, float val,
                               DLArrayHandle output) {
        /* TODO: Your code here */
        return 0;
}

int DLGpuMatrixMultiply(const DLArrayHandle matA, bool transposeA,
                        const DLArrayHandle matB, bool transposeB,
                        DLArrayHandle matC) {
        /* TODO: Your code here */
        // Hint: use cublas
        // cublas assume matrix is column major
        return 0;
}

int DLGpuRelu(const DLArrayHandle input, DLArrayHandle output) {
        /* TODO: Your code here */
        return 0;
}

int DLGpuReluGradient(const DLArrayHandle input, const DLArrayHandle in_grad,
                      DLArrayHandle output) {
        /* TODO: Your code here */
        return 0;
}

int DLGpuSoftmax(const DLArrayHandle input, DLArrayHandle output) {
        /* DONE: My code here */
        assert(input->ndim == 2);
        assert(output->ndim == 1);
        int nrow = input->shape[0];
        assert(nrow <= 1024 * 4);
        int ncol = input->shape[1];
        const float *input_data = (const float *)input->data;
        float *output_data = (float *)output->data;
        dim3 threads;
        threads.x = nrow % 1024;
        int nblocks = nrow / 1024;
        matrix_softmax_kernel<<<nblocks, threads>>>(
                nrow, ncol, input_data, output_data);
        return 0;
}

int DLGpuSoftmaxCrossEntropy(const DLArrayHandle input_a,
                             const DLArrayHandle input_b,
                             DLArrayHandle output) {
        assert(input_a->ndim == 2);
        assert(input_b->ndim == 2);
        assert(output->ndim == 1);
        assert(input_a->shape[0] == input_b->shape[0] &&
               input_a->shape[1] == input_b->shape[1]);
        int nrow = input_a->shape[0];
        // Maximum x- or y-dimension of a block = 1024
        // But we need 'nrow' shared memory, and max shared memory is 48KB.
        // Conservatively allow max 16KB shared memory.
        assert(nrow <= 1024 * 4);
        int ncol = input_a->shape[1];
        const float *input_data_a = (const float *)input_a->data;
        const float *input_data_b = (const float *)input_b->data;
        float *output_data = (float *)output->data;
        dim3 threads;
        threads.x = nrow % 1024;
        int nblocks = nrow / 1024;
        // 1 block, each block with 'threads' number of threads with 'nrow' shared
        // memory size
        matrix_softmax_cross_entropy_kernel<<<nblocks, threads >>>(
                nrow, ncol, input_data_a, input_data_b, output_data);
        return 0;
}
